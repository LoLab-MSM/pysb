#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>

#define num_species {n_species}
#define NPARAM {n_params}
#define NREACT {n_reactions}
#define NREACT_MIN_ONE NREACT-1




extern "C" {{

__device__ int stoch_matrix[]={{
{stoch}
}};

__device__ double sum_propensities(double *a){{
    double a0 = 0;
    #pragma unroll
    for(int j=0; j<NREACT; j++){{
        a0 += a[j];
    }}
    return a0;
}}

__device__ double propensities(int *y, double *h, double *param_vec)
{{
{hazards}
return sum_propensities(h);
}}


__device__ void stoichiometry(int *y, int r){{
    int step = r*num_species;
    for(int i=0; i<num_species; i++){{
        y[i]+=stoch_matrix[step + i];
    }}
}}



__device__ int sample(const double* a, double u){{
    int i = 0;
    #pragma unroll
    for(;i < NREACT_MIN_ONE && u > a[i]; i++){{
        u -= a[i];
        }}
    return i;
}}


__device__ void update_results(int* result, int *y,  int step, int time_index){{

    for(int j=0; j<num_species; j++){{
        result[step + j + (time_index * num_species)] = y[j];
    }}
}}

__global__ void Gillespie_all_steps(const int* species_matrix,  int* result,
                                    const double* time, const int NRESULTS,
                                    const double* param_values){{

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState randState;
//    hiprandStateMRG32k3a randState;
    hiprand_init(clock64(), tid, 0, &randState);


    int y[num_species];
    int prev[num_species];
    double A[NREACT] = {{0.0}};
    double param_vec[NPARAM] =  {{0.0}};
    const int result_stepping = tid*NRESULTS*num_species;

    // init parameters for thread
    #pragma unroll
    for(int i=0; i<NPARAM; i++){{
        param_vec[i] = param_values[tid*NPARAM + i];
        }}

    // init species counter for thread
    #pragma unroll
    for(int i=0; i<num_species; i++){{
        y[i] = species_matrix[tid*num_species + i];
        prev[i] = y[i];
        }}

    double t = time[0] ;
    int time_index = 0;
    // beginning of loop
    while (time_index < NRESULTS){{
        while (t < time[time_index]){{
            // store last state to be saved to output
            #pragma unroll
            for(int j=0; j<num_species; j++){{
                prev[j] = y[j];
            }}

            // calculate propensities
            double a0 = propensities(y, A, param_vec);
            if (a0 <= 0.0){{
                t = time[NRESULTS-1];
                continue;
            }}

            // calculate two random numbers
            double r1 =  hiprand_uniform(&randState);
            double r2 =  hiprand_uniform(&randState);

            // find time of next reaction and update time
            double tau = -__logf(r1)/a0;
            t += tau;

            // find next reaction and update species matrix
            double k = sample(A, a0*r2);
            stoichiometry(y, k);
            }}

        update_results(result, prev, result_stepping, time_index);
        time_index++;
        }}

    }}

}} // extern c close
